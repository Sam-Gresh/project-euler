#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
#include <time.h>
#include <stdint.h>

#define DEFAULT_THREADS_PER_BLOCK   256
#define DEFAULT_NUM_BLOCKS          10




__global__ void check_divis(unsigned long long max, uint_fast64_t* out, int thread_count){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint_fast64_t local_sum = 0;
    for(uint_fast64_t i = tid; i< max; i += thread_count){
        if (i % 3 == 0 || i % 5 == 0){
            local_sum += i;
        }
    }
    out[tid] = local_sum;
}


int main(int argc, char* argv[]) {
    uint_fast64_t* d_partial_sums;

    unsigned long long max;
    int threads_per_block;
    int numBlocks;
    int num_runs = 1;
    threads_per_block = DEFAULT_THREADS_PER_BLOCK;
    numBlocks = DEFAULT_NUM_BLOCKS;
    if(argc == 1){
        printf("Enter Maximum:\n");
        max = 1000;
        numBlocks = DEFAULT_NUM_BLOCKS;
        threads_per_block = DEFAULT_THREADS_PER_BLOCK;
    }
    if(argc >= 2){
        max = atoi(argv[1]);
    }
    if(argc >= 3){
        numBlocks = atoi(argv[2]);
    }
    if(argc >= 4){
        num_runs = atoi(argv[3]);
    }
    clock_t begin = clock();

    uint_fast64_t total;
    for(int run = 0; run < num_runs; run++){
        int thread_count = threads_per_block * numBlocks;

        hipMalloc((void**)&d_partial_sums, thread_count * sizeof(uint_fast64_t));



        check_divis<<<numBlocks, threads_per_block>>>(max, d_partial_sums, thread_count);

        uint_fast64_t* partial_sums = (uint_fast64_t*) malloc(thread_count * sizeof(uint_fast64_t));

        hipMemcpy(partial_sums, d_partial_sums, thread_count * sizeof(uint_fast64_t), hipMemcpyDeviceToHost);

        total = 0;
        for(int i = 0; i < thread_count; i++){
            total = total += partial_sums[i];
        }
        printf("The answer is: %llu\n", total);

        hipFree(d_partial_sums);
    }
    
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("The elapsed time is %f seconds\n", time_spent);
    return 0;
}