#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
#include <time.h>
#include <stdint.h>

#define DEFAULT_THREADS_PER_BLOCK   256
#define DEFAULT_NUM_BLOCKS          10




__global__ void check_divis(unsigned long long max, uint_fast64_t* out, int thread_count){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint_fast64_t local_sum = 0;
    for(uint_fast64_t i = tid; i< max; i += thread_count){
        if (i % 3 == 0 || i % 5 == 0){
            local_sum += i;
        }
    }
    out[tid] = local_sum;
}

__global__ void sum_list( uint_fast64_t* in, uint_fast64_t* out){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint_fast64_t local_sum = in[tid * 2] + in[tid * 2 + 1];
    out[tid] = local_sum;
}

int main(int argc, char* argv[]) {
    uint_fast64_t* d_partial_sums;
    uint_fast64_t* d_temp;
    unsigned long long max;
    
    int threads_per_block = DEFAULT_THREADS_PER_BLOCK;
    int num_blocks = DEFAULT_NUM_BLOCKS;
    int num_runs = 1;

    if(argc == 1){
        printf("Enter Maximum:\n");
        max = 1000;
        num_blocks = DEFAULT_NUM_BLOCKS;
        threads_per_block = DEFAULT_THREADS_PER_BLOCK;
    }
    if(argc >= 2){
        max = atoi(argv[1]);
    }
    if(argc >= 3){
        num_blocks = atoi(argv[2]);
    }
    if(argc >= 4){
        num_runs = atoi(argv[3]);
    }
    clock_t begin = clock();

    for(int run = 0; run < num_runs; run++){
        int thread_count = threads_per_block * num_blocks;
        int length = thread_count;
        


        hipMalloc((void**)&d_partial_sums, thread_count * sizeof(uint_fast64_t));
        hipMalloc((void**)&d_temp, thread_count * sizeof(uint_fast64_t));


        
        check_divis<<<num_blocks, threads_per_block>>>(max, d_partial_sums, thread_count);

        int nb = num_blocks;
        int tpb = threads_per_block;
        while(length > 1){
            
            sum_list<<<num_blocks, threads_per_block>>>(d_partial_sums, d_temp);
            if(nb > 1){
                nb /= 2;
            }
            else{
                tpb /= 2;
            }
            length /= 2;
            hipMemcpy(d_partial_sums, d_temp, length * sizeof(uint_fast64_t), hipMemcpyDeviceToDevice);
            
        }

        uint_fast64_t total;
        hipMemcpy(&total, d_partial_sums, 1 * sizeof(uint_fast64_t), hipMemcpyDeviceToHost);
        printf("The answer is: %llu\n", total);

        hipFree(d_partial_sums);
        hipFree(d_temp);
    }
    
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("The elapsed time is %f seconds\n", time_spent);
    return 0;
    
}